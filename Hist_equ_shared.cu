#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include "imageio.h"

//CUDA kernel for generating Histogram for gray Image(Naive Implementation)
__global__ void calculateHistogram(unsigned char *d_greyImage, int *d_histogram, int size)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    //printf("id:%d\n",id);
    if(id<size)
        atomicAdd(&(d_histogram[d_greyImage[id]]),1);
    __syncthreads();
}

//CUDA kernel for generating Histogram for gray Image(Stride based Implementation)
__global__ void calculateHistogramStride(unsigned char *d_greyImage, int *d_histogram, int size)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while(id < size)
    {
        atomicAdd(&(d_histogram[d_greyImage[id]]),1);
        id+=stride;
    }
}

__global__ void histogram_smem_atomics(unsigned char *d_greyImage, int *d_histogram, int imageWidth,int imageHeight)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;

    int t = threadIdx.x + threadIdx.y*blockDim.x;

    int nt = blockDim.x * blockDim.y;

    //int g = blockIdx.x + blockIdx.y *gridDim.x;

    __shared__ unsigned int smem[256];
    for(int i=t;i<256;i+=nt)
        smem[i] = 0;
    __syncthreads();
    for(int col=x;col<imageWidth;col+=nx)
    {
        for(int row=y; row<imageHeight;row+=ny)
        {
            atomicAdd(&smem[d_greyImage[row*imageWidth+col]],1);
        }
    }
    __syncthreads();
    for(int i = t;i<256;i+=nt)
        d_histogram[i] = smem[i];
}

//CUDA kernel for generating Histogram for gray Image(Shared Memory based Implementation)
__global__ void calculateHistogramShared(unsigned char *d_greyImage, int *d_histogram, int size)
{
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    while (id < size) {
        atomicAdd( &temp[d_greyImage[id]], 1);
        id += offset;
    }
    __syncthreads();
    atomicAdd(&(d_histogram[threadIdx.x]),temp[threadIdx.x] );
}

//Cuda kernel to apply histogram equalization method for image enhacement
__global__ void histogram_equalization(unsigned char *d_greyImage, int *d_histogram,unsigned char *d_enhanced, int size)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    //printf("id:%d\n",id);
    if(id<size)
        d_enhanced[id] = d_histogram[d_greyImage[id]];
    __syncthreads();
}

int main(int argc, char** argv)
{
    if(argc != 2){
        printf("Execute code in below format[Image must be in a ppm format]\n");
        printf("%s input_image\n", argv[0]);
        exit(-1);
    }
    
    PPMImage *image;
    char *input_image = argv[1];

    //Reading Image
    image = readPPM(input_image);
    int size = sizeof(unsigned char)*(image->x)*(image->y);
    int totalObservation = (image->x)*(image->y);

    //Memory Allocation and initialization of host variables
    unsigned char *h_rgb = (unsigned char *) malloc(size*3);
    unsigned char *h_greyImage = (unsigned char *) malloc(size);
    unsigned char *h_enhanced = (unsigned char *) malloc(size);
    unsigned char *d_greyImage;
    unsigned char *d_enhanced;
    unsigned int *h_histogram = (unsigned int *) malloc(sizeof(unsigned int)*256);
    int *d_histogram;

    //Memory Allocation of cuda variables 
    hipMalloc(&d_greyImage,size);
    hipMalloc(&d_enhanced,size);
    hipMalloc(&d_histogram,sizeof(int)*256);
    hipMemset(d_histogram, 0, 256*sizeof(int));

    //Cuda Variables to calculate execution time
    hipEvent_t start,stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //convert RGB image to grayscale for better image enhacement
    getGrayArray(image,h_rgb,h_greyImage);
    hipMemcpy(d_greyImage,h_greyImage,size,hipMemcpyHostToDevice);
    printf("Image Dimention: %dx%d pixels\n",imageWidth,imageHeight);

    //Required variables for executing CUDA kernel
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);
    int blocks = prop.multiProcessorCount;
    int blockSize, gridSize;
    blockSize = 1024;
    gridSize = (int)ceil((float)size/blockSize);

    hipEventRecord(start,0);
    calculateHistogramShared<<<blocks*2,256>>>(d_greyImage,d_histogram,size);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("Time Required for Creating Histogram: %3.5f ms\n",elapsedTime);

    hipMemcpy(h_histogram,d_histogram,sizeof(int) * 256, hipMemcpyDeviceToHost);

    //Calculating cummulative probabilities and new gray values for enhanced Image
    float cummulative = 0;
    for(int i=0;i<256;i++)
    {
        //printf("Cummulative Probability of gray image for value %d:%d\n",i,histogram[i]);
        cummulative = cummulative+(h_histogram[i]*1.0/totalObservation);
        h_histogram[i] = cummulative*255;
    }
    hipMemcpy(d_histogram,h_histogram,sizeof(int) * 256,hipMemcpyHostToDevice);
    //printf("Total Elements:%d",sum);

    hipEventRecord(start,0);
    histogram_equalization<<<gridSize,blockSize>>>(d_greyImage,d_histogram,d_enhanced,size);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("Total time required for Enhancing Image: %3.5f ms\n",elapsedTime);
    hipMemcpy(h_enhanced,d_enhanced,size,hipMemcpyDeviceToHost);

    //Create output contrast enhanced Image
    char *token = strtok(input_image, ".");
    const char *filename = strcat(token,".pgm");
    const int maxColorComponentValue = 255;
    FILE *fp;
    const char *comment = "# this is my new binary pgm file";
    fp = fopen(filename, "wb");
    // write header to the file 
    fprintf(fp, "P5\n %s\n %d\n %d\n %d\n", comment, imageWidth, imageHeight,maxColorComponentValue);
    // write image data bytes to the file
    fwrite(h_enhanced, imageHeight*imageWidth, 1, fp);
    fclose(fp);
    printf("OK - file %s saved\n", filename);

    //Deallocating Memories from host and device
    free(h_rgb);
    free(h_greyImage);
    free(h_histogram);
    free(h_enhanced);
    hipFree(d_greyImage);
    hipFree(d_histogram);
    hipFree(d_enhanced);
    return 0;
}